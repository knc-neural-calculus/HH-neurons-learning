#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/sequence.h>
#include <fstream>
#include <stdio.h>
#include <vector>

struct compute_san_output
{
    static constexpr float an_coupling_threshold = -60;
    static constexpr float Tmax = 1.0;
    static constexpr float C = 1;
    static constexpr float A = 0.02;
    static constexpr float vL = -60.95;
    static constexpr float vNa = 55;
    static constexpr float vK = -100;
    static constexpr float tauhA = 15;
    static constexpr float vCa = 120;
    static constexpr float kD = 30;
    static constexpr float vAMPA = 0;
    static constexpr float vNMDA = 0;
    static constexpr float vGABA = -70;
    static constexpr float alphaCa = 0.5;
    static constexpr float tauAMPA = 2;
    static constexpr float tausNMDA = 100;
    static constexpr float tauxNMDA = 2;
    static constexpr float tauGABA = 10;
    static constexpr float gL = 0.016307;
    static constexpr float gNa = 12.2438;
    static constexpr float gK = 19.20436;
    static constexpr float gA = 1.79259;
    static constexpr float gKS = 0.0350135;
    static constexpr float gNaP = 0.63314;
    static constexpr float gAR = 0.0166454;
    static constexpr float gCa = 0.1624;
    static constexpr float gKCa = 0.7506;
    static constexpr float gAMPA = 0.513425;
    static constexpr float gNMDA = 0.00434132;
    static constexpr float gGABA = 0.00252916;
    static constexpr float tauCa = 739.09;

    static constexpr float dt = 0.01;
    static constexpr float n_incs = 40000;

    __host__ __device__
        float operator()(float T)
    {
        // State variables.
        float V = 0.0, Ca = 0.1, nK = 0.0;

        // Intermediate variables.
        float mCa_inf, mKCa_inf, mNaP_inf,
            alpha_n, beta_n, ICa,
            dnKdt, dCadt, dvdt;

        float avg_out = 0.0;
        for (int i = 0; i < n_incs; ++i)
        {
            mCa_inf = 1 / (1 + expf(-(V + 20) / 9));
            mKCa_inf = 1 / (1 + powf(kD / Ca, 3.5));
            mNaP_inf = 1 / (1 + expf(-(V + 55.7) / 7.7));

            alpha_n = V == -34 ? 0.1 : 0.01 * (V + 34) / (1 - expf(-(V + 34) / 10));
            beta_n = 0.125 * expf(-(V + 44) / 25);

            // Calculate gating variable derivatives
            dnKdt = 4 * (alpha_n * (1 - nK) - beta_n * nK);

            // Calculate Ca2+ derivative
            ICa = gCa * mCa_inf * mCa_inf * (V - vCa);
            dCadt = -alphaCa * (10 * A * ICa) - Ca / tauCa;

            // Calculate voltage derivative
            dvdt = T
                - gK * powf(nK, 4) * (V - vK)
                - ICa
                - gKCa * mKCa_inf * (V - vK)
                - gNaP * mNaP_inf * mNaP_inf * mNaP_inf * (V - vNa)
                - gL * (V - vL);

            V += dt * dvdt;
            Ca += dt * dCadt;
            nK += dt * dnKdt;
            avg_out += 1 / (1 + expf(-(V + 55)/2.275));
        }
        return avg_out / n_incs;
    }

    static void record(float T, int n_incs_record, std::ostream& out)
    {
        // State variables.
        float V = 0.0, Ca = 0.1, nK = 0.0;

        // Intermediate variables.
        float mCa_inf, mKCa_inf, mNaP_inf,
            alpha_n, beta_n, ICa,
            dnKdt, dCadt, dvdt;

        for (int i = 0; i < n_incs_record; ++i)
        {
            mCa_inf = 1 / (1 + expf(-(V + 20) / 9));
            mKCa_inf = 1 / (1 + powf(kD / Ca, 3.5));
            mNaP_inf = 1 / (1 + expf(-(V + 55.7) / 7.7));

            alpha_n = V == -34 ? 0.1 : 0.01 * (V + 34) / (1 - expf(-(V + 34) / 10));
            beta_n = 0.125 * expf(-(V + 44) / 25);

            // Calculate gating variable derivatives
            dnKdt = 4 * (alpha_n * (1 - nK) - beta_n * nK);

            // Calculate Ca2+ derivative
            ICa = gCa * mCa_inf * mCa_inf * (V - vCa);
            dCadt = -alphaCa * (10 * A * ICa) - Ca / tauCa;

            // Calculate voltage derivative
            dvdt = T
                - gK * powf(nK, 4) * (V - vK)
                - ICa
                - gKCa * mKCa_inf * (V - vK)
                - gNaP * mNaP_inf * mNaP_inf * mNaP_inf * (V - vNa)
                - gL * (V - vL);

            V += dt * dvdt;
            Ca += dt * dCadt;
            nK += dt * dnKdt;
            out << dt * i << ',' << V << ',' << Ca << ',' << nK << std::endl;
        }
    }
};

struct compute_hh_output
{
    constexpr static const float gna = 120;  constexpr static const float ena = 55;                      // Sodium conductance and potential
    constexpr static const float gk = 36; constexpr static const float ek = -72;                         // Potassium conductance and potential
    constexpr static const float gl = 0.3; constexpr static const float el = -50;                        // Leak conductance and potential
    static constexpr float dt = 0.01;
    static constexpr float n_incs = 40000;

    __host__ __device__
        float operator()(float T)
    {
        // State variables.
        float V = 0.0, M = 0.0, N = 0.0, H = 1.0;

        // Intermediate variables.
        float Am, An, Ah, Bm, Bn, Bh,
              dvdt, dMdt, dNdt, dHdt;

        float avg_out = 0.0;
        for (int i = 0; i < n_incs; ++i)
        {
            // Calculate intermediate quantities
            Am = (3.5 + 0.1 * V) / (1 - exp(-3.5 - 0.1*V));
            An = (-0.5 - 0.01 * V) / (exp(-5 - 0.1 * V) - 1);
            Ah = 0.07 * exp(-V / 20 - 3);

            Bm = 4 * exp(-(V + 60) / 18);
            Bn = 0.125*exp(-(V + 60) / 80);
            Bh = 1 / (exp(-3 - 0.1*V) + 1);

            // Calculate gating variable derivatives
            dMdt = Am * (1 - M) - Bm * M;
            dNdt = An * (1 - N) - Bn * N;
            dHdt = Ah * (1 - H) - Bh * H;

            // Calculate voltage derivative
            dvdt = T
                - gna * M * M * M * H * (V - ena)
                - gk * powf(N, 4) * (V - ek)
                - gl * (V - el);

            V += dt * dvdt;
            M += dt * dMdt;
            N += dt * dNdt;
            H += dt * dHdt;
            avg_out += 1 / (1 + expf(-(V + 20)/3));
        }
        return avg_out / n_incs;
    }
};

int main()
{
    std::ofstream san_sample("san_sample.csv");
    compute_san_output::record(1, 2000 * 100, san_sample);
    san_sample.close();

    std::ofstream san_iv_plot("san_iv_plot_no_zoom.csv");
    int N = 1e6;
    thrust::device_vector<float> output(N), current(N);
    thrust::sequence(current.begin(), current.end(), 0.0f, 5.0f / N);
    thrust::transform(current.begin(), current.end(), output.begin(), compute_san_output());
    thrust::host_vector<float> output_cpu(output), current_cpu(current);
    for (int i = 0; i < N; ++i)
        san_iv_plot << current_cpu[i] << ',' << output_cpu[i] << std::endl;
    san_iv_plot.close();

    san_iv_plot.open("san_iv_plot_zoom_1.csv");
    thrust::sequence(current.begin(), current.end(), 2.4f, 0.7f / N);
    thrust::transform(current.begin(), current.end(), output.begin(), compute_san_output());
    output_cpu = output;
    current_cpu = current;
    for (int i = 0; i < N; ++i)
        san_iv_plot << current_cpu[i] << ',' << output_cpu[i] << std::endl;
    san_iv_plot.close();

    san_iv_plot.open("san_iv_plot_zoom_2.csv");
    thrust::sequence(current.begin(), current.end(), 2.7f, 0.1f / N);
    thrust::transform(current.begin(), current.end(), output.begin(), compute_san_output());
    output_cpu = output;
    current_cpu = current;
    for (int i = 0; i < N; ++i)
        san_iv_plot << current_cpu[i] << ',' << output_cpu[i] << std::endl;
    san_iv_plot.close();

    std::ofstream hh_iv_plot("hh_iv_plot.csv");
    thrust::sequence(current.begin(), current.end(), 5.0f, 10.0f / N);
    thrust::transform(current.begin(), current.end(), output.begin(), compute_hh_output());
    output_cpu = output;
    current_cpu = current;
    for (int i = 0; i < N; ++i)
        hh_iv_plot << current_cpu[i] << ',' << output_cpu[i] << std::endl;
    hh_iv_plot.close();
}
